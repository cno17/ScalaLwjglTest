
#include <hip/hip_runtime.h>

extern "C" __global__ void add(int n, float *a, float *b, float *res) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        res[i] = a[i] + b[i];
    }
}
